#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "glyphs/device/GlyphsGeom.h"
#include "glyphs/device/PerRayData.h"
#include "glyphs/device/RayGenData.h"
#include "glyphs/device/Camera.h"
#include "glyphs/device/disney_bsdf.h"
#include "glyphs/device/TriangleMesh.h"

namespace glyphs {
  namespace device {

    inline __device__
    int32_t make_8bit(const float f)
    {
      return min(255,max(0,int(f*256.f)));
    }
    
    inline __device__
    int32_t make_rgba8(const vec4f color)
    {
      return
        (make_8bit(color.x) << 0) +
        (make_8bit(color.y) << 8) +
        (make_8bit(color.z) << 16);
    }

    inline __device__ vec3f random_in_unit_sphere(Random &rnd) {
      vec3f p;
      do {
        p = 2.0f*vec3f(rnd(),rnd(),rnd()) - vec3f(1.f, 1.f, 1.f);
      } while (dot(p,p) >= 1.0f);
      return p;
    }
    
    // ------------------------------------------------------------------
    // A simple path tracer; if pathDepth <= 1 falls back to local
    // shading.
    // Enable FAST_SHADING via cmake option to get Lambertian
    // instead of Disney BRDF
    // ------------------------------------------------------------------

    inline __device__
    vec3f missColor(const Ray &ray)
    {
      const vec2i pixelID = owl::getLaunchIndex();
      const float t = pixelID.y / (float)optixGetLaunchDimensions().y;
      const vec3f c = (1.0f - t)*vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
      return c;
    }

    inline __device__
    vec3f pathTrace(const RayGenData &self,
                    owl::Ray &ray,
                    Random &rnd,
                    PerRayData &prd)
    {
      vec3f attenuation = 1.f;
      vec3f ambientLight(.8f);

      const FrameState *fs = &self.frameStateBuffer[0];
      int pathDepth = fs->pathDepth;
      
      if (pathDepth <= 1) {
        prd.primID = -1;
        owl::traceRay(/*accel to trace against*/self.world,
                      /*the ray to trace*/ ray,
                      /*prd*/prd/*,
                              OPTIX_RAY_FLAG_DISABLE_ANYHIT*/);

        if (prd.primID < 0)
          return missColor(ray);
        
        vec3f N = prd.Ng;
        if (dot(N,(vec3f)ray.direction)  > 0.f)
          N = -N;
        N = normalize(N);
        
        vec3f albedo;
        
        // Random colors for glyphs, grey for triangles
        if (prd.meshID == 0) {
          albedo = vec3f(.8f);
        } else {
          unsigned rgba = self.linkBuffer[prd.primID].col; // ignore alpha for now
          albedo = vec3f((rgba & 0xff) / 255.f,
                        ((rgba >> 8) & 0xff) / 255.f,
                        ((rgba >> 16) & 0xff) / 255.f);
        }
        vec3f color = albedo * (.2f+.6f*fabsf(dot(N,(vec3f)ray.direction)));
        return color;
      }

      // could actually swtich material based on meshID ...
      DisneyMaterial material = fs->material;
      /* iterative version of recursion, up to depth 50 */
      for (int depth=0;true;depth++) {
        prd.primID = -1;
        owl::traceRay(/*accel to trace against*/self.world,
                      /*the ray to trace*/ ray,
                      /*prd*/prd/*,
                      OPTIX_RAY_FLAG_DISABLE_ANYHIT*/);
        
        if (prd.primID == -1) {
          // miss...
          if (depth == 0)
            return missColor(ray);

#if FAST_SHADING
          return attenuation * ambientLight;
#else
          float phi = atan2(ray.direction.y, ray.direction.x);
          float theta = acos(ray.direction.z / length(ray.direction));
          const float half_width = 0.1f;

          if (theta > (0.55f - half_width) * M_PIF && theta < (0.55f + half_width) * M_PIF
              && phi > (0.75f - half_width) * M_PIF && phi < (0.75f + half_width) * M_PIF) {
            return attenuation * owl::vec3f(8.f);
          } else {
            return attenuation * owl::vec3f(ambientLight / 2.f);
          }
#endif
        }

        vec3f N = normalize(prd.Ng);
        const vec3f w_o = -ray.direction;
        if (dot(N, w_o) < 0.f) {
          N = -N;
        }
        
        // Random colors for glyphs, grey for triangles
        if (prd.meshID >= 0) { 
          material.base_color = vec3f(.8f);
        } else {
          unsigned rgba = self.linkBuffer[prd.primID].col; // ignore alpha for now
          material.base_color = vec3f((rgba & 0xff) / 255.f,
                                      ((rgba >> 8) & 0xff) / 255.f,
                                      ((rgba >> 16) & 0xff) / 255.f);
        }

        owl::vec3f v_x, v_y;
        ortho_basis(v_x, v_y, N);
        // pdf and dir are set by sampling the BRDF
        float pdf;
        vec3f scattered_direction;
        vec3f albedo = sample_disney_brdf(material, N, w_o, v_x, v_y, rnd,
                                          scattered_direction, pdf);
        
        const vec3f scattered_origin    = ray.origin + prd.t * ray.direction;
        ray = owl::Ray(/* origin   : */ scattered_origin,
                       /* direction: */ scattered_direction,
                       /* tmin     : */ 1e-3f,
                       /* tmax     : */ 1e+8f);

        if (depth >= pathDepth || pdf == 0.f || albedo == owl::vec3f(0.f)) {
          // ambient term:
          return owl::vec3f(0.f);//attenuation * ambientLight;
        }

        attenuation *= albedo * fabs(dot(scattered_direction, N)) / pdf;
      }
    }

    OPTIX_MISS_PROGRAM(miss_program)()
    {
      /*! nothing to do - we initialize prd before trace */
    }

    /*! the actual ray generation program - note this has no formal
      function parameters, but gets its paramters throught the 'pixelID'
      and 'pixelBuffer' variables/buffers declared above */
    OPTIX_RAYGEN_PROGRAM(raygen_program)()
    {
      const RayGenData &self = owl::getProgramData<RayGenData>();
      const vec2i pixelID = owl::getLaunchIndex();
      const vec2i launchDim = owl::getLaunchDims();
  
      if (pixelID.x >= self.fbSize.x) return;
      if (pixelID.y >= self.fbSize.y) return;
      const int pixelIdx = pixelID.x+self.fbSize.x*pixelID.y;

      // for multi-gpu: only render every deviceCount'th column of 32 pixels:
      if (((pixelID.x/32) % self.deviceCount) != self.deviceIndex)
        return;
      
      uint64_t clock_begin = clock64();
      const FrameState *fs = &self.frameStateBuffer[0];
      int pixel_index = pixelID.y * launchDim.x + pixelID.x;
      vec4f col(0.f);
      Random rnd(pixel_index,
                 fs->accumID
                 );

      PerRayData prd;
      prd.rnd = &rnd;

      for (int s = 0; s < fs->samplesPerPixel; s++) {
        vec2f pixelSample = vec2f(pixelID) + vec2f(rnd(),rnd());
        float u = float(pixelID.x + rnd());
        float v = float(pixelID.y + rnd());
        owl::Ray ray = Camera::generateRay(*fs, pixelSample, rnd);
        col += vec4f(pathTrace(self,ray,rnd,prd),1);
      }
      col = col / float(fs->samplesPerPixel);

      uint64_t clock_end = clock64();
      if (fs->heatMapEnabled) {
        float t = (clock_end-clock_begin)*fs->heatMapScale;
        if (t >= 256.f*256.f*256.f)
          col = vec4f(1,0,0,1);
        else {
          uint64_t ti = uint64_t(t);
          col.x = ((ti >> 16) & 255)/255.f;
          col.y = ((ti >> 8) & 255)/255.f;
          col.z = ((ti >> 0) & 255)/255.f;
        }
      }
    
      if (fs->accumID > 0)
        col = col + (vec4f)self.accumBufferPtr[pixelIdx];
      self.accumBufferPtr[pixelIdx] = col;

      uint32_t rgba = make_rgba8(col / (fs->accumID+1.f));
      self.colorBufferPtr[pixelIdx] = rgba;
    }
 
  }
}

