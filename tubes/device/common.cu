#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "tubes/device/TubesGeom.h"
#include "tubes/device/PerRayData.h"
#include "tubes/device/RayGenData.h"
#include "tubes/device/Camera.h"

#include <optix_device.h> // Only for test, can be remove later. 

namespace tubes {
  namespace device {
    


    inline __device__
    int32_t make_8bit(const float f)
    {
      return min(255,max(0,int(f*256.f)));
    }
    
    inline __device__
    int32_t make_rgba8(const vec4f color)
    {
      return
        (make_8bit(color.x) << 0) +
        (make_8bit(color.y) << 8) +
        (make_8bit(color.z) << 16);
    }

    inline __device__ vec3f random_in_unit_sphere(Random &rnd) {
      vec3f p;
      do {
        p = 2.0f*vec3f(rnd(),rnd(),rnd()) - vec3f(1, 1, 1);
      } while (dot(p,p) >= 1.0f);
      return p;
    }
    
    // ------------------------------------------------------------------
    // actual tubes stuff
    // ------------------------------------------------------------------

    /*
      OPTIX_INTERSECT_PROGRAM(tubes_intersect)()
      {
      const int primID = optixGetPrimitiveIndex();
      const auto &self
      = owl::getProgramData<TubesGeom>();
      
      owl::Ray ray(optixGetWorldRayOrigin(),
      optixGetWorldRayDirection(),
      optixGetRayTmin(),
      optixGetRayTmax());

      const Link link = self.links[primID];
      float tmp_hit_t = ray.tmax;
      if (intersectSphere(link,self.radius,ray,tmp_hit_t)) {
      optixReportIntersection(tmp_hit_t,primID);
      }
      }
    */
    


    inline __device__
    vec3f missColor(const Ray &ray)
    {
      const vec2i pixelID = owl::getLaunchIndex();
      const float t = pixelID.y / (float)optixGetLaunchDimensions().y;
      const vec3f c = (1.0f - t)*vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
      return c;
    }

    
    
    inline __device__ vec3f traceRay(const RayGenData &self,
                                     owl::Ray &ray,
                                     Random &rnd,
                                     PerRayData &prd)
    {
      vec3f attenuation = 1.f;
      vec3f ambientLight(.8f);

      const FrameState *fs = &self.frameStateBuffer[0];
      int pathDepth = fs->shadeMode;
      /* code for tubes */
      if (pathDepth <= 1) {
        prd.instID = -1;
        // prd.numIsecs = 0;
        owl::traceRay(/*accel to trace against*/self.world,
                      /*the ray to trace*/ ray,
                      /*prd*/prd);

        // if (owl::getLaunchIndex()*2 == owl::getLaunchDims())
        //   printf("isecs %i\n",prd.numIsecs);
        if (prd.instID < 0)
          return missColor(ray);

        const Arrow arrow = self.arrowBuffer[prd.instID];

        vec3f N = prd.gn;

        if (dot(N,(vec3f)ray.direction)  > 0.f)
          N = -N;
        N = normalize(N);

        // hardcoded albedo for now:
        const vec3f albedo// = vec3f(0.628, 0.85, 0.511);
        = randomColor(1+prd.instID);
        vec3f color = albedo * (.2f+.6f*fabsf(dot(N,(vec3f)ray.direction)));
        return color;
      }


      
      /* iterative version of recursion, up to depth 50 */
      for (int depth=0;true;depth++) {
        prd.instID = -1;
        owl::trace(/*accel to trace against*/self.world,
                   /*the ray to trace*/ ray,
                   /*numRayTypes*/1,
                   /*prd*/prd,
                   0);
        // rtTrace(world, ray, prd, RT_VISIBILITY_ALL, RT_RAY_FLAG_DISABLE_ANYHIT);
        
        if (prd.instID == -1) {
          // miss...
          if (depth == 0)
            return missColor(ray);
          return attenuation * ambientLight;
        }
        
        const Arrow arrow = self.arrowBuffer[prd.instID];
        vec3f N = prd.gn;
        // Normal of the cylinder. No more on-the-fly calculation needed.        
        // printf("normal %f %f %f\n",N.x,N.y,N.z);

        //if (dot(N,(vec3f)ray.direction)  > 0.f)
        //  N = -N;
        
        N = normalize(N);

        // hardcoded albedo for now:
        const vec3f albedo = vec3f(.6f);
        // = randomColor(1+prd.instID);//link.matID);
        // hard-coded for the 'no path tracing' case:
        // if (pathDepth <= 1)
        //   return albedo * (.2f+.6f*fabsf(dot(N,(vec3f)ray.direction)));
          
        attenuation *= albedo;
        //attenuation *=  (.2f + .6f * fabsf(dot(N, (vec3f)ray.direction)));

        if (depth >= pathDepth) {
          // ambient term:
          return attenuation * ambientLight;
        }
        
        const vec3f scattered_origin    = ray.origin + prd.t * ray.direction;
        const vec3f scattered_direction = N + random_in_unit_sphere(rnd);
        ray = owl::Ray(/* origin   : */ scattered_origin,
                       /* direction: */ normalize(scattered_direction),
                       /* tmin     : */ 1e-3f,
                       /* tmax     : */ 1e+8f);
      }
    }

    OPTIX_MISS_PROGRAM(miss_program)()
    {
      /*! nothing to do - we initialize prd before trace */
    }

    /*! the actual ray generation program - note this has no formal
      function parameters, but gets its paramters throught the 'pixelID'
      and 'pixelBuffer' variables/buffers declared above */
    OPTIX_RAYGEN_PROGRAM(raygen_program)()
    {
      const RayGenData &self = owl::getProgramData<RayGenData>();
      const vec2i pixelID = owl::getLaunchIndex();
      const vec2i launchDim = owl::getLaunchDims();
  
      if (pixelID.x >= self.fbSize.x) return;
      if (pixelID.y >= self.fbSize.y) return;
      const int pixelIdx = pixelID.x+self.fbSize.x*pixelID.y;

      // for multi-gpu: only render every deviceCount'th column of 32 pixels:
      if (((pixelID.x/32) % self.deviceCount) != self.deviceIndex)
        return;
      
      uint64_t clock_begin = clock64();
      const FrameState *fs = &self.frameStateBuffer[0];
      int pixel_index = pixelID.y * launchDim.x + pixelID.x;
      vec4f col(0.f);
      Random rnd(pixel_index,
                 fs->accumID
                 );

      PerRayData prd;

      for (int s = 0; s < fs->samplesPerPixel; s++) {
        vec2f pixelSample = vec2f(pixelID) + vec2f(rnd(),rnd());
        float u = float(pixelID.x + rnd());
        float v = float(pixelID.y + rnd());
        owl::Ray ray = Camera::generateRay(*fs, pixelSample, rnd);
        col += vec4f(traceRay(self,ray,rnd,prd),1);
      }
      col = col / float(fs->samplesPerPixel);

      uint64_t clock_end = clock64();
      if (fs->heatMapEnabled) {
        float t = (clock_end-clock_begin)*fs->heatMapScale;
        if (t >= 256.f*256.f*256.f)
          col = vec4f(1,0,0,1);
        else {
          uint64_t ti = uint64_t(t);
          col.x = ((ti >> 16) & 255)/255.f;
          col.y = ((ti >> 8) & 255)/255.f;
          col.z = ((ti >> 0) & 255)/255.f;
        }
      }
    
      if (fs->accumID > 0)
        col = col + (vec4f)self.accumBufferPtr[pixelIdx];
      self.accumBufferPtr[pixelIdx] = col;

      uint32_t rgba = make_rgba8(col / (fs->accumID+1.f));
      self.colorBufferPtr[pixelIdx] = rgba;
    }
    
  }
}

