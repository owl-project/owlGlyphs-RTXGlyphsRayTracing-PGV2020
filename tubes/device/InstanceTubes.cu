#include "hip/hip_runtime.h"
#include <float.h>
#include "tubes/device/TubesGeom.h"
#include "tubes/device/PerRayData.h"
#include "tubes/device/RayGenData.h"
#include "tubes/device/Camera.h"

namespace tubes {
  namespace device {

    inline __device__
    float sign(float& val)
    {
      return val < 0.0f ? -1.0f : 1.0;
    }

    inline __device__
    int32_t make_8bit(const float f)
    {
      return min(255, max(0, int(f * 256.f)));
    }

    inline __device__
    int32_t make_rgba8(const vec4f color)
    {
      return
        (make_8bit(color.x) << 0) +
        (make_8bit(color.y) << 8) +
        (make_8bit(color.z) << 16);
    }

    inline __device__
    vec3f uIntToVec3f(int32_t irgba)
    {
      vec3f rgb;
      rgb.z = uint_as_float(irgba >> 16 & 0xFF) / 255;
      rgb.y = uint_as_float(irgba >> 8 & 0xFF) / 255;
      rgb.x = uint_as_float(irgba & 0xFF) / 255;
      return rgb;
    }

    inline __device__
    int32_t vec3fToUInt(vec3f t)
    {
      t.x = min(0.f, max(t.x, 1.0f));   // clamp to [0.0, 1.0]
      t.y = min(0.f, max(t.y, 1.0f));
      t.z = min(0.f, max(t.z, 1.0f));
      return (int32_t(t.z * 255) << 16) | (int32_t(t.y * 255) << 8) | int32_t(t.x * 255);
    }

    inline __device__ vec3f random_in_unit_sphere(Random& rnd) {
      vec3f p;
      do {
        p = 2.0f * vec3f(rnd(), rnd(), rnd()) - vec3f(1, 1, 1);
      } while (dot(p, p) >= 1.0f);
      return p;
    }

    // ------------------------------------------------------------------
    // actual arrows stuff
    // ------------------------------------------------------------------

    /*! ray-cylinder intersector : ref code from shadertoy.com/view/4lcSRn */
    inline __device__ bool intersectCylinder(const owl::Ray ray, const Cylinder &cylinder, float &hit_t, vec3f &isec_normal)
    {
      const float ra = cylinder.radius;
      const vec3f  pa = cylinder.p1;
      const vec3f  pb = cylinder.p2;

      const vec3f  ba = pb - pa;
      const vec3f  oc = ray.origin - pa;

      float baba = dot(ba, ba);
      float bard = dot(ba, ray.direction);
      float baoc = dot(ba, oc);

      float k2 = baba - bard * bard;
      float k1 = baba * dot(oc, ray.direction) - baoc * bard;
      float k0 = baba * dot(oc, oc) - baoc * baoc - ra * ra * baba;

      float h = k1 * k1 - k2 * k0;

      if (h < 0.f) return false;

      h = sqrtf(h);
      float t = (-k1 - h) / k2;

      // body
      float y = baoc + t * bard;
      if (y > 0.0 && y < baba) {
        hit_t = t;
        isec_normal = (oc + t * ray.direction - ba * y / baba) / ra;
        return true;
      }
      // caps notice we only need one side 
      t = (((y < 0.0) ? 0.0 : baba) - baoc) / bard;
      if (abs(k1 + k2 * t) < h)
        {
          hit_t = t;
          isec_normal = ba * sign(y) / baba;
          return true;
        }
      return false;
    }

    inline __device__ bool intersectCone(const owl::Ray ray, const Cone &cone, float &hit_t, vec3f &n)
    {
      const vec3f dir(ray.direction);

      vec3f co(ray.origin.x - cone.pos.x,
               ray.origin.z - cone.pos.z,
               cone.height - ray.origin.y + cone.pos.y);
      float A = (cone.radius / cone.height) * (cone.radius / cone.height);

      float a = dir.x*dir.x + dir.z*dir.z - A*dir.y*dir.y;
      float b = 2.f*co.x*dir.x + 2.f*co.y*dir.z + 2.f*A*co.z*dir.y;
      float c = co.x*co.x + co.y*co.y - A*co.z*co.z;

      float det = b*b - 4.f*a*c;
      if (det < 0.f) return false;

      det = sqrtf(det);
      float t1 = (-b - det) / (2.f * a);
      float t2 = (-b + det) / (2.f * a);

      float t = (t1<=t2)?t1:t2;
      float h = ray.origin.y+t*dir.y;

      if (h>cone.pos.y && h<cone.pos.y+cone.height) {
        hit_t = t;
        vec3f p = ray.origin+ray.direction*t;
        float r = sqrtf((p.x-cone.pos.x)*(p.x-cone.pos.x)+(p.z-cone.pos.z)*(p.z-cone.pos.z));
        n = vec3f(p.x-cone.pos.x, r*(cone.radius/cone.height), p.z-cone.pos.z);
        return true;
      } else {
        return false;
      }
    }

    inline __device__ bool intersectArrow(const owl::Ray ray, const Arrow &arrow, float &hit_t, vec3f &n)
    {
      bool result = false;

      ArrowParts parts = disassemble(arrow);

      // Test against cylinder
      float cyl_t = FLT_MAX;
      vec3f cyl_n;
      bool hit_cyl = intersectCylinder(ray, parts.cyl, cyl_t, cyl_n);
      if (hit_cyl && cyl_t<hit_t) {
        hit_t = cyl_t;
        n = cyl_n;
        result = true;
      }

      // Test against cone
      float cone_t = FLT_MAX;
      vec3f cone_n;
      bool hit_cone = intersectCone(ray, parts.cone, cone_t, cone_n);
      if (hit_cone && cone_t<hit_t) {
        hit_t = cone_t;
        n = cone_n;
        result = true;
      }

      return result;
    }

    //Correct the intersection program.
    OPTIX_INTERSECT_PROGRAM(Arrows_intersect)()
    {
      const int instID = optixGetPrimitiveIndex();
      const auto& self
        = owl::getProgramData<ArrowsGeom>();
      //owl::Ray ray(optixGetWorldRayOrigin(),
      //             optixGetWorldRayDirection(),
      //             optixGetRayTmin(),
      //             optixGetRayTmax());
      owl::Ray ray(optixGetObjectRayOrigin(),
          optixGetObjectRayDirection(),
          optixGetRayTmin(),
          optixGetRayTmax());

      const Arrow arrow = self.arrows[instID];
      float t = ray.tmax;
      vec3f normal;

      if (intersectArrow(ray, arrow, t, normal)) {
        if(optixReportIntersection(t,instID)) {
          PerRayData &prd = owl::getPRD<PerRayData>();
          prd.instID = instID;
          prd.t = t;
          prd.gn = normal;
        }
      }
    }

    // Round Cone boundingBox
    OPTIX_BOUNDS_PROGRAM(Arrows_bounds)(const void* geomData,
                                        box3f& primBounds,
                                        const int    instID)
    {
      const ArrowsGeom& self = *(const ArrowsGeom*)geomData;
      const Arrow& arrow = self.arrows[instID];
      primBounds = get_bounds(arrow);
    }

    OPTIX_CLOSEST_HIT_PROGRAM(Arrows_closest_hit)()
    {  }
  }
}
