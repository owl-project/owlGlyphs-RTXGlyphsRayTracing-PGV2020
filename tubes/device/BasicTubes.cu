#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "tubes/device/TubesGeom.h"
#include "tubes/device/PerRayData.h"
#include "tubes/device/RayGenData.h"
#include "tubes/device/Camera.h"

namespace tubes {
  namespace device {
    
    inline __device__
    float sign(float & val)
    {
      return val < 0.0f ? -1.0f : 1.0;
    }


    inline __device__
    bool intersectSphere2(const vec3f   pa,
                          const float   ra,
                          const owl::Ray ray,
                          float& hit_t,
                          vec3f& isec_normal)
    {
// #if STATS_ON
//       STATS_COUNT_LINK_TEST();
// #endif
      const vec3f  oc = ray.origin - pa;
      const float  a = dot((vec3f)ray.direction, (vec3f)ray.direction);
      const float  b = dot(oc, (vec3f)ray.direction);
      const float  c = dot(oc, oc) - ra * ra;
      const float  discriminant = b * b - a * c;

      if (discriminant < 0.f) return false;

      {
        float temp = (-b - sqrtf(discriminant)) / a;
        if (temp < hit_t && temp > ray.tmin) {
          hit_t = temp;
          isec_normal = ray.origin + hit_t * ray.direction - pa;
          return true;
        }
      }

      {
        float temp = (-b + sqrtf(discriminant)) / a;
        if (temp < hit_t && temp > ray.tmin) {
          hit_t = temp;
          isec_normal = ray.origin + hit_t * ray.direction - pa;
          return true;
        }
      }
      return false;
    }


    //Correct the intersection program.
    /*! ray-cylinder intersector : ref code from shadertoy.com/view/4lcSRn */
    inline __device__ bool intersectCylinder(const vec3f   pa,
                                             const vec3f   pb,
                                             const float   ra,
                                             const owl::Ray ray,
                                             float &hit_t,
                                             vec3f &isec_normal)
    {
#if STATS_ON
      STATS_COUNT_LINK_TEST();
#endif
      const vec3f  ba = pb - pa;  
      const vec3f  oc = ray.origin - pa;

      float baba = dot(ba,ba);
      float bard = dot(ba,ray.direction);
      float baoc = dot(ba,oc);

      float k2 = baba                       - bard*bard;
      float k1 = baba*dot(oc,ray.direction) - baoc*bard;
      float k0 = baba*dot(oc,oc)            - baoc*baoc - ra*ra*baba;
    
      float h = k1*k1 - k2*k0;
   
      if (h < 0.f) return false;
      
      h = sqrtf(h);
      float t = (-k1-h)/k2;
      
      // body
      float y = baoc + t*bard;
      if( y>0.0 && y<baba ){
        hit_t = t;
        isec_normal = (oc+t*ray.direction - ba*y/baba)/ra;
        return true;
      }

      // caps
      t = ( ((y<0.0) ? 0.0 : baba) - baoc)/bard;
      if( abs(k1+k2*t)<h )
        {
          hit_t = t;
          isec_normal =  ba*sign(y)/baba;
          return true;
        }
      return false;
    }



    /* ray - rounded cone intersection. */
    inline __device__
        bool intersectRoundedCone(
            const vec3f  pa, const vec3f  pb,
            const float  ra, const float  rb,
            const owl::Ray ray,
            float& hit_t,
            vec3f& isec_normal)
    {
        const vec3f& ro = ray.origin;
        const vec3f& rd = ray.direction;

        vec3f  ba = pb - pa;
        vec3f  oa = ro - pa;
        vec3f  ob = ro - pb;
        float  rr = ra - rb;
        float  m0 = dot(ba, ba);
        float  m1 = dot(ba, oa);
        float  m2 = dot(ba, rd);
        float  m3 = dot(rd, oa);
        float  m5 = dot(oa, oa);
        float  m6 = dot(ob, rd);
        float  m7 = dot(ob, ob);

        float d2 = m0 - rr * rr;

        float k2 = d2 - m2 * m2;
        float k1 = d2 * m3 - m1 * m2 + m2 * rr * ra;
        float k0 = d2 * m5 - m1 * m1 + m1 * rr * ra * 2.0 - m0 * ra * ra;

        float h = k1 * k1 - k0 * k2;
        if (h < 0.0) return false;
        float t = (-sqrtf(h) - k1) / k2;

        float y = m1 - ra * rr + t * m2;
        if (y > 0.0 && y < d2)
        {
            hit_t = t;
            isec_normal = normalize(d2 * (oa + t * rd) - ba * y);
            return true;
        }

        // Caps. 
        float h1 = m3 * m3 - m5 + ra * ra;
        if (h1 > 0.0)
        {
            t = -m3 - sqrtf(h1);
            hit_t = t;
            isec_normal = normalize((oa + t * rd) / ra);
            return true;
        }
        return false;
    }
    OPTIX_INTERSECT_PROGRAM(basicTubes_intersect)()
    {
        const int primID = optixGetPrimitiveIndex();
        const auto& self
            = owl::getProgramData<TubesGeom>();

        owl::Ray ray(optixGetWorldRayOrigin(),
            optixGetWorldRayDirection(),
            optixGetRayTmin(),
            optixGetRayTmax());
        const Link link = self.links[primID];
        if (link.prev < 0) return;

        float tmp_hit_t = ray.tmax;

        vec3f pb, pa; float ra, rb;
        pa = link.pos;
        ra = link.rad;
        if (link.prev >= 0) {
            rb = self.links[link.prev].rad;
            pb = self.links[link.prev].pos;
            vec3f normal;

            if (intersectRoundedCone(pa, pb, ra,rb, ray, tmp_hit_t, normal))
            {
                if (optixReportIntersection(tmp_hit_t, primID)) {
                    PerRayData& prd = owl::getPRD<PerRayData>();
                    prd.linkID = primID;
                    prd.t = tmp_hit_t;
                    prd.isec_normal = normal;
                }
            }
        }
    }

    OPTIX_INTERSECT_PROGRAM(tubes_intersect)()
    {
      const int primID =  optixGetPrimitiveIndex();
      const auto &self
        = owl::getProgramData<TubesGeom>();
        
      owl::Ray ray(optixGetWorldRayOrigin(),
                   optixGetWorldRayDirection(),
                   optixGetRayTmin(),
                   optixGetRayTmax());
      const Link link = self.links[primID];
      if(link.prev<0) return;

      float tmp_hit_t = ray.tmax;

      vec3f pb,pa; //float ra,rb;        
      pa = link.pos;               
      // ra = link.rad;

      PerRayData& prd = owl::getPRD<PerRayData>();
      vec3f normal;
      
      if (intersectSphere2(pa,self.radius,ray,
                           tmp_hit_t, normal)) {
        if (optixReportIntersection(tmp_hit_t, 1)) {
          prd.linkID = primID;
          prd.t = tmp_hit_t;
          prd.isec_normal = normal;
        }
      } else
        tmp_hit_t = ray.tmax;
      
      if (link.prev >= 0) {
        // rb = self.links[link.prev].rad;
        pb = self.links[link.prev].pos;        

        // prd.numIsecs++;
        if (intersectCylinder(pa,pb,self.radius,ray,tmp_hit_t,normal))       
          {
            if(optixReportIntersection(tmp_hit_t,primID)) {
              prd.linkID = primID;
              prd.t = tmp_hit_t;
              prd.isec_normal = normal;
            }
          }
      }
    }
    
    // Original Sphere boundingbox
    /*OPTIX_BOUNDS_PROGRAM(tubes_bounds)(const void  *geomData,
      box3f       &primBounds,
      const int    primID)
      {
      const TubesGeom &self = *(const TubesGeom*)geomData;
      const Link &link = self.links[primID];
      primBounds = box3f()
      .including(link.pos-self.radius)
      .including(link.pos+self.radius);
      }*/

      // Round Cone boundingBox
    OPTIX_BOUNDS_PROGRAM(basicTubes_bounds)(const void* geomData,
        box3f& primBounds,
        const int    primID)
    {
        const TubesGeom& self = *(const TubesGeom*)geomData;
        const Link& link = self.links[primID];

        vec3f pa = link.pos;
        float ra = link.rad;

        float rb = link.prev < 0 ? ra : self.links[link.prev].rad;
        vec3f pb = link.prev < 0 ? pa : self.links[link.prev].pos;

        vec3f a = pb - pa;
        vec3f ee = vec3f(1.0) - a * a / dot(a, a);
        vec3f e = ra * vec3f(sqrtf(ee.x), sqrtf(ee.y), sqrtf(ee.z));

        primBounds = box3f()
            .including(min(min(pa - e * ra, pb - e * rb), min(pa - ra, pb - rb)))  // be carefull with this.
            .including(max(max(pa + e * ra, pb + e * rb), max(pa + ra, pb + rb))); //
    }

    // Cylinder boundingBox    
    OPTIX_BOUNDS_PROGRAM(tubes_bounds)(const void  *geomData,
                                       box3f       &primBounds,
                                       const int    primID)
    {
      const TubesGeom &self = *(const TubesGeom*)geomData;
      const Link &link = self.links[primID];

      //vec3f pa = link.pos;                      //Not sure about the order of the links.
      //vec3f pb = self.links[link.prev].pos;
      //float ra = self.radius ;
      
      vec3f pa = link.pos;
      // float ra = link.rad;
      primBounds
        = box3f()
        .including(pa-self.radius)
        .including(pa+self.radius);
      // primBounds
      //   = box3f()
      //   .including(pa-ra)
      //   .including(pa+ra);

      if (link.prev >= 0) {
        // add space that connects to previous point:
        // float rb = self.links[link.prev].rad;
        vec3f pb = self.links[link.prev].pos;
        primBounds
          = primBounds
          .including(pb-self.radius)
          .including(pb+self.radius);
        // primBounds
        //   = primBounds
        //   .including(pb-rb)
        //   .including(pb+rb);
      }
    }

    OPTIX_CLOSEST_HIT_PROGRAM(tubes_closest_hit)()
    {
      // const float x = __uint_as_float(optixGetAttribute_0());
      // const float y = __uint_as_float(optixGetAttribute_1());
      // const float z = __uint_as_float(optixGetAttribute_2());
      
      // const int primID = optixGetPrimitiveIndex();
      // PerRayData &prd  = owl::getPRD<PerRayData>();
                        
      // prd.linkID       = primID;
      // prd.t            = optixGetRayTmax();
      // prd.isec_normal = vec3f(x,y,z);
    }    
  }
}

